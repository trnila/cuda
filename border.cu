#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "check.h"

__global__ void border(uchar3* array, int w, int h) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if(x >= w || y >= h) {
		return;
	}

	if(x % 20 == 0 || x % 20 == 1 || y % 20 == 0 || y % 20 == 1) {
		array[y * w + x] = (uchar3) {255, 255, 255};
	}
}

void cuda_border(uchar3 *img, int w, int h) {
	uchar3 *cuda = NULL;
	checkErr(hipMalloc(&cuda, sizeof(uchar3) * w * h));
	checkErr(hipMemcpy(cuda, img, sizeof(uchar3)*w*h, hipMemcpyHostToDevice));

	int block = 20;
	dim3 blocks((w+block) / block, h/(block));
	dim3 threads(block, block);

	border<<<blocks, threads>>>(cuda, w, h);
	checkErr(hipPeekAtLastError());
	checkErr(hipMemcpy(img, cuda, sizeof(uchar3)*w*h, hipMemcpyDeviceToHost));
	checkErr(hipFree(cuda));
}


