#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "check.h"

__global__ void kernel(uchar3 *orig, uchar3 *resized, int w, int h, int w1, int h1) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if(x >= w1 || y >= h1) {
		return;
	}

	uchar3 px1 = orig[y*2 * w + x*2];
	uchar3 px2 = orig[(y*2 + 1) * w + x*2 + 1];

	resized[y * w1 + x].x = (px1.x + px2.x) / 2;
	resized[y * w1 + x].y = (px1.y + px2.y) / 2;
	resized[y * w1 + x].z = (px1.z + px2.z) / 2;
}

uchar3* cuda_resize(uchar3 *img, int w, int h, int w1, int h1) {
	uchar3 *dst = new uchar3[w1 * h1];

	uchar3 *orig = NULL;
	uchar3 *resized = NULL;
	checkErr(hipMalloc(&orig, sizeof(uchar3) * w * h));
	checkErr(hipMalloc(&resized, sizeof(uchar3) * w1 * h1));

	checkErr(hipMemcpy(orig, img, sizeof(uchar3) * w * h, hipMemcpyHostToDevice));

	int count = 10;
	dim3 blocks((w1 + count)/ count, (h1 + count) / count);
	dim3 threads(count, count);
	kernel<<<blocks, threads>>>(orig, resized, w, h, w1, h1);
	checkErr(hipPeekAtLastError());
	checkErr(hipMemcpy(dst, resized, sizeof(uchar3)*w1*h1, hipMemcpyDeviceToHost));
	checkErr(hipFree(orig));
	checkErr(hipFree(resized));

	return dst;
}