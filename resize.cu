#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "check.h"


#include <stdio.h>
__global__ void kernel(uchar4 *orig, uchar4 *resized, int w, int h) {
	int x = blockDim.x * blockIdx.x;
	int y = blockDim.y * blockIdx.y;

	uchar4 a = {0, 0, 0, 0};
	int vx = 0;
	int vy = 0;
	int vz = 0;

	for(int i = x; i < x + 100; i++) {
		for(int j = y; j < y + 100; j++) {
			vx += orig[j * w + i].x;
			vy += orig[j * w + i].y;
			vz += orig[j * w + i].z;

			//a.x = (a.x + orig[j * w + i].x) % 255;
			//a.y = (a.y + orig[j * w + i].y) % 255;
			//a.z = (a.z + orig[j * w + i].z) % 255;
		}
	}

	//a.x = b / 100;
	//a.x = (vx / 100) % 255;
	//a.y = vy / 100;
	//a.z = vz / 100;
	a.x = 255;

	printf("%d\n", a.x);

	resized[y * w + x] = a;
}

void cuda_resize(uchar4 *img, uchar4 *dst, int w, int h, int w1, int h1) {
	uchar4 *orig = NULL;
	uchar4 *resized = NULL;
	checkErr(hipMalloc(&orig, sizeof(uchar4) * w * h));
	checkErr(hipMalloc(&resized, sizeof(uchar4) * w1 * h1));

	int count = 10;
	dim3 blocks(w / count, h / count);
	dim3 threads(count, count);
	kernel<<<blocks, threads>>>(orig, resized, w1, h1);
	checkErr(hipPeekAtLastError());
	checkErr(hipMemcpy(dst, resized, sizeof(uchar4)*w1*h1, hipMemcpyDeviceToHost));
	checkErr(hipFree(orig));
	checkErr(hipFree(resized));

}