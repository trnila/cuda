#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "check.h"

__global__ void rotate(uchar4* array, int w, int h) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if(x >= w || y >= h) {
		return;
	}

	uchar4 a = array[y * w + x];
	array[y * w + x] = array[(h - y - 1) * w + x];
	array[(h - y - 1) * w + x] = a;
}

void cuda_rotate(uchar4 *img, int w, int h) {
	uchar4 *cuda = NULL;
	checkErr(hipMalloc(&cuda, sizeof(uchar4) * w * h));
	checkErr(hipMemcpy(cuda, img, sizeof(uchar4)*w*h, hipMemcpyHostToDevice));

	int block = 20;
	dim3 blocks((w+block) / block, h/(2*block));
	dim3 threads(block, block);

	rotate<<<blocks, threads>>>(cuda, w, h);
	checkErr(hipPeekAtLastError());
	checkErr(hipMemcpy(img, cuda, sizeof(uchar4)*w*h, hipMemcpyDeviceToHost));
	checkErr(hipFree(cuda));
}


