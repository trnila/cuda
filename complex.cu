#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "check.h"

__global__ void fkernel(uchar3 *orig, uchar3 *resized, int w, int h) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if(x >= w || y >= h) {
		return;
	}


	int sx = w / 2;
	int sy = h / 2;

	float theta = 30 * 3.14 / 180;
	int x2 = (x-sx) * cos(theta) - (y - sy) * sin(theta) + sx;
	int y2 = (x-sx) * sin(theta) + (y - sy) * cos(theta) + sy;

	if(x2 >= 0 && x2 < w && y2 >=0 && y2 < h) {
		resized[y * w + x] = orig[y2 * w + x2];
	} else {
		resized[y * w + x] = (uchar3) {0, 0, 0};
	}
}

uchar3* cuda_slope(uchar3 *img, int w, int h) {
	uchar3 *dst = new uchar3[w * h];

	uchar3 *orig = NULL;
	uchar3 *resized = NULL;
	checkErr(hipMalloc(&orig, sizeof(uchar3) * w * h));
	checkErr(hipMalloc(&resized, sizeof(uchar3) * w * h));

	checkErr(hipMemcpy(orig, img, sizeof(uchar3) * w * h, hipMemcpyHostToDevice));

	int count = 10;
	dim3 blocks((w + count)/ count, (h + count) / count);
	dim3 threads(count, count);
	fkernel<<<blocks, threads>>>(orig, resized, w, h);
	checkErr(hipPeekAtLastError());
	checkErr(hipMemcpy(dst, resized, sizeof(uchar3)*w*h, hipMemcpyDeviceToHost));
	checkErr(hipFree(orig));
	checkErr(hipFree(resized));

	return dst;
}