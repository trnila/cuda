#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>

#define checkErr(result) check(result, __FILE__, __LINE__)

void check(hipError_t err, const char *file, int line) {
	if(err != hipSuccess) {
		fprintf(stderr, "Error: %s in %s:%d\n", hipGetErrorString(err), file, line);
		exit(1);
	}
}

__global__ void test(uchar4* array, int w, int h) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if(x >= w || y >= h) {
		return;
	}

	uchar4 a = array[y * w + x];
	array[y * w + x] = array[(h - y - 1) * w + x];
	array[(h - y - 1) * w + x] = a;
}

void run_cuda(uchar4 *img, int w, int h) {
	uchar4 *cuda = NULL;
	checkErr(hipMalloc(&cuda, sizeof(uchar4) * w * h));
	checkErr(hipMemcpy(cuda, img, sizeof(uchar4)*w*h, hipMemcpyHostToDevice));

	int block = 20;
	dim3 blocks((w+block) / block, h/(2*block));
	dim3 threads(block, block);

	test<<<blocks, threads>>>(cuda, w, h);
	checkErr(hipPeekAtLastError());
	checkErr(hipMemcpy(img, cuda, sizeof(uchar4)*w*h, hipMemcpyDeviceToHost));
	checkErr(hipFree(cuda));
}


