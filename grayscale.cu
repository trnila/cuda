#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "check.h"

__global__ void grayscale(uchar3* array, int w, int h) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if(x >= w || y >= h) {
		return;
	}

	uchar3 px = array[y * w + x];
	px.z = px.y = px.x = px.x * 0.114 + px.y * 0.587 + px.z * 0.299;
	array[y * w + x] = px;
}

void cuda_grayscale(uchar3 *img, int w, int h) {
	uchar3 *cuda = NULL;
	checkErr(hipMalloc(&cuda, sizeof(uchar3) * w * h));
	checkErr(hipMemcpy(cuda, img, sizeof(uchar3)*w*h, hipMemcpyHostToDevice));

	int block = 20;
	dim3 blocks((w+block) / block, h/(block));
	dim3 threads(block, block);

	grayscale<<<blocks, threads>>>(cuda, w, h);
	checkErr(hipPeekAtLastError());
	checkErr(hipMemcpy(img, cuda, sizeof(uchar3)*w*h, hipMemcpyDeviceToHost));
	checkErr(hipFree(cuda));
}


