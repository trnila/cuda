#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "check.h"

__global__ void grayscale(uchar4* array, int w, int h) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if(x >= w || y >= h) {
		return;
	}

	uchar4 px = array[y * w + x];
	px.z = px.y = px.x = px.x * 0.114 + px.y * 0.587 + px.z * 0.299;
	array[y * w + x] = px;
}

void cuda_grayscale(uchar4 *img, int w, int h) {
	uchar4 *cuda = NULL;
	checkErr(hipMalloc(&cuda, sizeof(uchar4) * w * h));
	checkErr(hipMemcpy(cuda, img, sizeof(uchar4)*w*h, hipMemcpyHostToDevice));

	int block = 20;
	dim3 blocks((w+block) / block, h/(block));
	dim3 threads(block, block);

	grayscale<<<blocks, threads>>>(cuda, w, h);
	checkErr(hipPeekAtLastError());
	checkErr(hipMemcpy(img, cuda, sizeof(uchar4)*w*h, hipMemcpyDeviceToHost));
	checkErr(hipFree(cuda));
}


